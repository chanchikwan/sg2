#include "hip/hip_runtime.h"
#include "ihd.h"

/* Compute dF/dx and dF/dy from W */
static __global__ void _dx_dd_dy_dd(C *x, C *y, const C *w,
                                    const Z n1, const Z h2)
{
  const Z i = blockDim.y * blockIdx.y + threadIdx.y;
  const Z j = blockDim.x * blockIdx.x + threadIdx.x;
  const Z h = i * h2 + j;

  if(i < n1 && j < h2) {
    const C u = w[h];
    R lx = i < n1 / 2 ? i : i - n1;
    R ly = j;

    if(h) {
      const R ikk = K(1.0) / (lx * lx + ly * ly);
      lx *= ikk;
      ly *= ikk;
    }

    x[h].r = - lx * u.i;
    x[h].i =   lx * u.r;
    y[h].r = - ly * u.i;
    y[h].i =   ly * u.r;
  }
}

void dx_dd_dy_dd(C *x, C *y, C *w)
{
  _dx_dd_dy_dd<<<Hsz, Bsz>>>(X, Y, W, N1, H2);
}

/* Compute dF/dx and dW/dy from W for the 1st term in the Jacobian */
static __global__ void _dx_dd_dy(C *x, C *y, const C *w,
                                 const Z n1, const Z h2)
{
  const Z i = blockDim.y * blockIdx.y + threadIdx.y;
  const Z j = blockDim.x * blockIdx.x + threadIdx.x;
  const Z h = i * h2 + j;

  if(i < n1 && j < h2) {
    const C u  = w[h];
    const R kx = i < n1 / 2 ? i : i - n1;
    const R ky = j;

    R lx = kx;
    if(h) lx *= K(1.0) / (kx * kx + ky * ky);

    x[h].r = - lx * u.i;
    x[h].i =   lx * u.r;
    y[h].r = - ky * u.i;
    y[h].i =   ky * u.r;
  }
}

void dx_dd_dy(C *x, C *y, C *w)
{
  _dx_dd_dy<<<Hsz, Bsz>>>(x, y, w, N1, H2);
}

/* Compute dF/dy and dW/dx from W for the 2nd term in the Jacobian */
static __global__ void _dy_dd_dx(C *y, C *x, const C *w,
                                 const Z n1, const Z h2)
{
  const Z i = blockDim.y * blockIdx.y + threadIdx.y;
  const Z j = blockDim.x * blockIdx.x + threadIdx.x;
  const Z h = i * h2 + j;

  if(i < n1 && j < h2) {
    const C u  = w[h];
    const R kx = i < n1 / 2 ? i : i - n1;
    const R ky = j;

    R ly = ky;
    if(h) ly *= K(1.0) / (kx * kx + ky * ky);

    y[h].r = - ly * u.i;
    y[h].i =   ly * u.r;
    x[h].r = - kx * u.i;
    x[h].i =   kx * u.r;
  }
}

void dy_dd_dx(C *y, C *x, C *w)
{
  _dy_dd_dx<<<Hsz, Bsz>>>(y, x, w, N1, H2);
}
