#include "hip/hip_runtime.h"
#include <stdio.h>
#include "ihd.h"

#define TIDE 512

static __global__ void _reduce(C *out, const R *x, const R *y,
                                       const Z n1, const Z n2, const Z N2)
{
  __shared__ R max[TIDE], sum[TIDE];

  const Z t = threadIdx.x;
  const Z j = blockDim.x * blockIdx.x + t;
  Z i;

  max[t] = 0.0;
  sum[t] = 0.0;

  if(j < n2) for(i = 0; i < n1; ++i) {
    const Z h  = i * N2 + j;
    const R ux = x[h];
    const R uy = y[h];
    const R uu = ux * ux + uy * uy;
    if(max[t] < uu) max[t] = uu;
    sum[t] += uu;
  }
  __syncthreads();

  for(i = blockDim.x / 2; i > 0; i /= 2) {
    if(t < i) {
      if(max[t] < max[t + i]) max[t] = max[t + i];
      sum[t] += sum[t + i];
    }
    __syncthreads();
  }

  if(t == 0) {
    out[blockIdx.x].r = max[0];
    out[blockIdx.x].i = sum[0];
  }
}

static FILE *file = NULL;

static void close(void)
{
  if(file) fclose(file);
}

R diag(void)
{
  const Z bsz = TIDE;
  const Z gsz = (N2 - 1) / bsz + 1;

  Z i;
  R max = 0.0, sum = 0.0;

  dx_dd_dy_dd(X, Y, W);
  inverse((R *)X, X);
  inverse((R *)Y, Y);
  _reduce<<<gsz, bsz>>>((C *)w, (R *)X, (R *)Y, N1, N2, F2);

  hipMemcpy(Host, w, sizeof(C) * gsz, hipMemcpyDeviceToHost);

  for(i = 0; i < gsz; ++i) {
    if(max < Host[i].r) max = Host[i].r;
    sum += Host[i].i;
  }

  if(!file) {
    atexit(close);
    file = fopen("log", "w");
  }

  fprintf(file, "%g\n", 0.5 * sum / (N1 * N2));
  fflush(file);

  return sqrt(max);
}
