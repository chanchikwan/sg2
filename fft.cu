#include <hipfft/hipfft.h>
#include "sg2.h"

#define CONCATENATION(PREFIX, NAME) PREFIX ## NAME
#define CONCATE_MACRO(PREFIX, NAME) CONCATENATION(PREFIX, NAME)

#define FAIL(X, ...) (HIPFFT_SUCCESS != CONCATE_MACRO(cufft, X)(__VA_ARGS__))

#if defined(DOUBLE) || defined(OUBLE) /* So -DOUBLE works */
#  define HIPFFT_R2C    HIPFFT_D2Z
#  define HIPFFT_C2R    HIPFFT_Z2D
#  define ExecR2C      ExecD2Z
#  define ExecC2R      ExecZ2D
#  define hipfftReal    hipfftDoubleReal
#  define hipfftComplex hipfftDoubleComplex
#endif

static hipfftHandle r2c, c2r;

void mkplans(Z n1, Z n2)
{
  if(FAIL(Plan2d, &r2c, n1, n2, HIPFFT_R2C) ||
     FAIL(Plan2d, &c2r, n1, n2, HIPFFT_C2R) ||
     FAIL(SetCompatibilityMode, r2c, CUFFT_COMPATIBILITY_FFTW_PADDING) ||
     FAIL(SetCompatibilityMode, c2r, CUFFT_COMPATIBILITY_FFTW_PADDING))
    error("CUFFT ERROR :\tfail to create plan(s).\n");
}

void rmplans(void)
{
  if(FAIL(Destroy, r2c) ||
     FAIL(Destroy, c2r))
    error("CUFFT ERROR :\tfail to destroy plan(s).\n");
}

C *forward(C *F, R *f)
{
  if(FAIL(ExecR2C, r2c, (hipfftReal *)f, (hipfftComplex *)F))
    error("CUFFT ERROR :\tfail to perform forward transform.\n");
  return F;
}

R *inverse(R *f, C *F)
{
  if(FAIL(ExecC2R, c2r, (hipfftComplex *)F, (hipfftReal *)f))
    error("CUFFT ERROR :\tfail to perform inverse transform.\n");
  return f;
}
