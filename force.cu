#include "hip/hip_runtime.h"
#include <stdlib.h>
#include "ihd.h"

/* We turn on Kolmogorov forcing when fi * ki < 0.  Therefore, there
   is an extra minus sign in line 19 although we are using

     f_K = fi ki cos(ki x)

   as the forcing. */
static __global__ void _force(R *f, const R sl, const R fi, const R ki,
                                    const Z N1, const Z N2)
{
  const Z i = blockDim.y * blockIdx.y + threadIdx.y;
  const Z j = blockDim.x * blockIdx.x + threadIdx.x;
  const Z h = i * N2 + j;

  if(i < N1 && j < N2) {
    const R dx = K(1.0) / N1;
    f[h] = sl * f[h] - fi * ki * cos(TWO_PI * ki * i * dx);
  }
}

R *force(R *f, R sl, R fi, R ki)
{
  _force<<<Gsz, Bsz>>>(f, sl, fi, ki, N1, N2);
  return f;
}

/* When ky != 0, the random forcing is straightforward. */
static __global__ void _force1(C *F, const R fx, const R fy,
                                     const Z kx, const Z ky,
                                     const Z N1, const Z H2)
{
  const Z i = blockDim.y * blockIdx.y + threadIdx.y;
  const Z j = blockDim.x * blockIdx.x + threadIdx.x;
  const Z h = i * H2 + j;

  if(i < N1 && j < H2) {
    const Z k = i < N1 / 2 ? i : i - N1;
    if(k == kx && (j == ky || j == -ky)) {
      F[h].r += fx;
      F[h].i += fy;
    }
  }
}

/* After transforming along the y-direction, the ky == 0 column is
   real.  It is necessary to implement the Hermit symmetric by hand.
   The force is added to both the k == kx and k ==-kx modes. */
static __global__ void _force2(C *F, const R fx, const R fy,
                                     const Z kx, const Z ky,
                                     const Z N1, const Z H2)
{
  const Z i = blockDim.y * blockIdx.y + threadIdx.y;
  const Z j = blockDim.x * blockIdx.x + threadIdx.x;
  const Z h = i * H2 + j;

  if(i < N1 && j < H2) {
    const Z k = i < N1 / 2 ? i : i - N1;
    if(k == kx && j == ky) {
      F[h].r += fx;
      F[h].i += fy;
    }
    if(k ==-kx && j == ky) {
      F[h].r += fx;
      F[h].i -= fy;
    }
  }
}

C *force(C *F, R dt, R fi, R ki)
{
  const R fs = fi * ki * sqrt(dt); /* no factor of 2 because of FFT */
  const R dp = TWO_PI / (RAND_MAX + 1.0);
  const R pm = dp * (Seed = rand());
  const R pk = dp * (Seed = rand());

  const R fx = fs * cos(pm);
  const R fy = fs * sin(pm);
  const Z kx = (Z)floor(ki * cos(pk) + 0.5);
  const Z ky = (Z)floor(ki * sin(pk) + 0.5);

  if(ky) _force1<<<Hsz, Bsz>>>(F, fx, fy, kx, ky, N1, H2);
  else   _force2<<<Hsz, Bsz>>>(F, fx, fy, kx, ky, N1, H2);

  return F;
}
