#include <stdio.h>
#include "ihd.h"

R *load(R *f, const char *name)
{
  FILE *file;
  Z     size[3];

  file = fopen(name, "rb");
  fread(size, sizeof(Z), 3,       file); /* TODO: check data type and size */
  fread(Host, sizeof(R), N1 * N2, file);
  fclose(file);

  hipMemcpy(f, Host, sizeof(R) * N1 * N2, hipMemcpyHostToDevice);

  return f;
}

R *dump(const char *name, R *f)
{
  FILE *file;
  Z     size[3] = {sizeof(R), N1, N2};

  hipMemcpy(Host, f, sizeof(R) * N1 * N2, hipMemcpyDeviceToHost);

  file = fopen(name, "wb");
  fwrite(size, sizeof(Z), 3,       file);
  fwrite(Host, sizeof(R), N1 * N2, file);
  fclose(file);

  return f;
}
