#include <stdio.h>
#include "ihd.h"

R *load(R *f, Z i)
{
  char  name[256];
  FILE *file;
  Z     size[3];

  sprintf(name, "%04d.raw", i);
  file = fopen(name, "rb");
  fread(size, sizeof(Z), 3,       file); /* TODO: check data type and size */
  fread(Host, sizeof(R), N1 * N2, file);
  fclose(file);

  hipMemcpy(f, Host, sizeof(R) * N1 * N2, hipMemcpyHostToDevice);

  return f;
}

Z dump(Z i, R *f)
{
  char  name[256];
  FILE *file;
  Z     size[3] = {sizeof(R), N1, N2};

  hipMemcpy(Host, f, sizeof(R) * N1 * N2, hipMemcpyDeviceToHost);

  sprintf(name, "%04d.raw", i);
  file = fopen(name, "wb");
  fwrite(size, sizeof(Z), 3,       file);
  fwrite(Host, sizeof(R), N1 * N2, file);
  fclose(file);

  return i;
}
