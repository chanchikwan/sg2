#include <stdio.h>
#include "ihd.h"

R *load(R *f, Z i)
{
  char  name[256];
  FILE *file;

  sprintf(name, "%04d.raw", i);
  file = fopen(name, "rb");
  fread(Host, sizeof(R), N1 * N2, file);
  fclose(file);

  hipMemcpy(f, Host, sizeof(R) * N1 * N2, hipMemcpyHostToDevice);

  return f;
}

Z dump(Z i, R *f)
{
  char  name[256];
  FILE *file;

  hipMemcpy(Host, f, sizeof(R) * N1 * N2, hipMemcpyDeviceToHost);

  sprintf(name, "%04d.raw", i);
  file = fopen(name, "wb");
  fwrite(Host, sizeof(R), N1 * N2, file);
  fclose(file);

  return i;
}
