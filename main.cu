#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "ihd.h"

R noise(R x, R y)
{
  return 0.5 - (R)rand() / RAND_MAX;
}

R decay(R x, R y)
{
  return 1024.0 * noise(x, y); /* so u = curl(w) ~ 1 */
}

R KH(R x, R y)
{
  return noise(x, y) + (fabs(x - 0.0 ) < 1.0e-6 ? -512.0 : 0.0)
                     + (fabs(x - M_PI) < 1.0e-6 ?  512.0 : 0.0);
}

int main(int argc, char *argv[])
{
  const char rotor[] = "-/|\\";
  const char input[] = "input.raw";

  R nu = (argc > 1) ? atof(argv[1]) : 1.0e-5;
  R mu = (argc > 2) ? atof(argv[2]) : 1.0e-2;
  R fi = (argc > 3) ? atof(argv[3]) : 5.0e-2;
  R ki = (argc > 4) ? atof(argv[4]) : 1.0e+2;
  R tt = (argc > 5) ? atof(argv[5]) : 1024.0;

  Z n0 = (argc > 6) ? atoi(argv[6]) : 1024;
  Z n1 = (argc > 7) ? atoi(argv[7]) : 1024;
  Z n2 = (argc > 8) ? atoi(argv[8]) : 1024;

  R fo = 5 * n1 * n2 * (21.5 + (fi * ki > 0.0 ? 0 : 8) +
                        12.5 * (log2((double)n1) + log2((double)n2)));
  Z i  = 0;

  hipEvent_t t0, t1;
  hipEventCreate(&t0);
  hipEventCreate(&t1);

  printf("2D spectral hydrodynamic code with CUDA\n");
  setup(n1, n2);

  if(exist(input))
    scale(forward(W, load(w, input)), 1.0 / (n1 * n2));
  else {
    scale(forward(W, init(w, noise)), 1.0 / (n1 * n2));
    dump(name(i), inverse(w, W));
  }

  while(i++ < n0) {
    float ms;
    Z ns = (Z)ceil(tt / n0 / 0.9 / getdt(1.0, nu, mu)), j;
    R dt =         tt / n0 / ns;
    printf("%4d: %5.2f -> %5.2f, dt ~ %.0e:       ",
           i, dt * ns * (i-1), dt * ns * i, dt);

    hipEventRecord(t0, 0);
    for(j = 0; j < ns; ++j) {
      printf("\b\b\b\b\b\b%c %4d", rotor[j%4], j+1);
      fflush(stdout);
      step(nu, mu, fi, ki, dt);
    }
    hipEventRecord(t1, 0);

    hipEventSynchronize(t1);
    hipEventElapsedTime(&ms, t0, t1); ms /= ns;
    printf("\b\b\b\b\b\b%.3f ms/cycle ~ %.3f GFLOPS\n",
           ms, 1e-6 * fo / ms);

    dump(name(i), inverse(w, W));
  }

  hipEventDestroy(t1);
  hipEventDestroy(t0);

  return 0;
}
