#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "ihd.h"

R noise(R x, R y)
{
  return 0.5 - (R)rand() / RAND_MAX;
}

R decay(R x, R y)
{
  return 1024.0 * noise(x, y); /* so u = curl(w) ~ 1 */
}

R KH(R x, R y)
{
  return noise(x, y) + (fabs(x - 0.0 ) < 1.0e-6 ? -512.0 : 0.0)
                     + (fabs(x - M_PI) < 1.0e-6 ?  512.0 : 0.0);
}

int main(int argc, char *argv[])
{
  const char rotor[] = "-/|\\";

  R nu = (argc > 1) ? atof(argv[1]) : 1.0e-4;
  R mu = (argc > 2) ? atof(argv[2]) : 1.0e-2;
  R tt = (argc > 3) ? atof(argv[3]) : 1.0e+2;

  Z n0 = (argc > 4) ? atoi(argv[4]) : 1024;
  Z n1 = (argc > 5) ? atoi(argv[5]) : 1024;
  Z n2 = (argc > 6) ? atoi(argv[6]) : 1024;

  R fo = 5 * n1 * n2 * (21.5 + 12.5 * (log2((double)n1) + log2((double)n2)));
  Z i  = 0;

  hipEvent_t t0, t1;
  hipEventCreate(&t0);
  hipEventCreate(&t1);

  printf("2D spectral hydrodynamic code with CUDA\n");
  setup(n1, n2);

  scale(forward(W, init(w, KH)), 1.0 / (n1 * n2));
  dump(i, inverse(w, W));

  while(i++ < n0) {
    float ms;
    Z ns = (Z)ceil(tt / n0 / 0.9 / getdt(10.0, nu, mu)), j;
    R dt =         tt / n0 / ns;
    printf("%4d: %5.2f -> %5.2f, dt ~ %.0e:       ",
           i, dt * ns * (i-1), dt * ns * i, dt);

    hipEventRecord(t0, 0);
    for(j = 0; j < ns; ++j) {
      printf("\b\b\b\b\b\b%c %4d", rotor[j%4], j+1);
      fflush(stdout);
      step(nu, mu, dt);
    }
    hipEventRecord(t1, 0);

    hipEventSynchronize(t1);
    hipEventElapsedTime(&ms, t0, t1); ms /= ns;
    printf("\b\b\b\b\b\b%.3f ms/cycle ~ %.3f GFLOPS\n",
           ms, 1e-6 * fo / ms);

    dump(i, inverse(w, W));
  }

  hipEventDestroy(t1);
  hipEventDestroy(t0);

  return 0;
}
