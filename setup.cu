#include "hip/hip_runtime.h"
#include <stdlib.h>
#include "sg2.h"

#define TIDE 32

Z N1, N2, H2, F2;
uint3 Bsz, Gsz, Hsz;

R *w, Flop;
C *W, *X, *Y, *Host;

static void done(void)
{
  rmplans();

  hipFree(Y);
  hipFree(X);
  hipFree(W);
  hipFree(w);

  free(Host);
}

void setup(Z n1, Z n2)
{
  hipDeviceProp_t dev;
  Z m;

  atexit(done);

  hipGetDeviceProperties(&dev, 0);
  m = dev.maxThreadsPerBlock;

  N1 = n1;
  N2 = n2;
  H2 = n2 / 2 + 1; /* number of non-redundant coefficients */
  F2 = H2 * 2;     /* necessary for in-place transform     */

  Bsz = make_uint3(TIDE, m / TIDE, 1);
  Gsz = make_uint3((N2 - 1) / Bsz.x + 1, (N1 - 1) / Bsz.y + 1, 1);
  Hsz = make_uint3((H2 - 1) / Bsz.x + 1, (N1 - 1) / Bsz.y + 1, 1);

  Host = (C *)malloc(sizeof(C) * N1 * H2);

  hipMalloc(&w, sizeof(R) * N1 * F2); scale((C *)w, 0.0);
  hipMalloc(&W, sizeof(C) * N1 * H2); scale((C *)W, 0.0);
  hipMalloc(&X, sizeof(C) * N1 * H2); scale((C *)X, 0.0);
  hipMalloc(&Y, sizeof(C) * N1 * H2); scale((C *)Y, 0.0);

  mkplans(n1, n2);
}
