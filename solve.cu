#include <stdlib.h>
#include <stdio.h>
#include "sg2.h"

int solve(R nu, R mu, R fi, R ki, R dT, Z i, Z n)
{
  const char rotor[] = "-/|\\";

  R time = dT * i;

  hipEvent_t t0, t1;
  hipEventCreate(&t0);
  hipEventCreate(&t1);

  printf("======================= Start Simulation =======================\n");

  while(++i <= n) {
    const R next = dT * i;
    Z m = 0;
    float ms;
    printf("%4d: %5.2f -> %5.2f:                  ", i, time, next);

    hipEventRecord(t0, 0);
    while(time < next) {
      R dt = getdt(nu, mu, fi);
      if(dt == 0.0) error(" diverged, QUIT\n");
      printf("\b\b\b\b\b\b\b\b\b\b\b\b\b\b\b\b\b\b%5d ", ++m);
      if(time + dt < next)
        time += dt;
      else {
        dt = next - time;
        time = next;
      }
      printf("%c dt ~ %5.0e", rotor[m%4], dt);
      fflush(stdout);
      step(nu, mu, fi, ki, dt);
    }
    hipEventRecord(t1, 0);

    hipEventSynchronize(t1);
    hipEventElapsedTime(&ms, t0, t1); ms /= m;
    printf("\b\b\b\b\b\b\b\b\b\b\b\bstep%c %7.3f ms/cycle ~ %.3f GFLOPS\n",
           m > 1 ? 's' : ' ', ms, 1e-6 * flop() / ms);

    dump(name(i), W);
  }

  printf("======================= Done  Simulation =======================\n");

  hipEventDestroy(t1);
  hipEventDestroy(t0);
  return 0;
}
