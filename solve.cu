#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "ihd.h"

static R typical_v = 1.0;

int solve(R nu, R mu, R fi, R ki, R tt, Z i, Z n)
{
  const char rotor[] = "-/|\\";
  const R flop = 5 * N1 * N2 * (21.5 + (fi * ki > 0.0 ? 0 : 8) +
                                12.5 * (log2((double)N1) + log2((double)N2)));
  hipEvent_t t0, t1;
  hipEventCreate(&t0);
  hipEventCreate(&t1);

  printf("======================= Start Simulation =======================\n");

  for(++i; i <= n; ++i) {
    float ms;
    Z ns = (Z)ceil(tt / n / getdt(typical_v, nu, mu)), j;
    R dt =         tt / n / ns;
    printf("%4d: %5.2f -> %5.2f, dt ~ %.0e:       ",
           i, dt * ns * (i-1), dt * ns * i, dt);
    srand(Seed);

    hipEventRecord(t0, 0);
    for(j = 0; j < ns; ++j) {
      printf("\b\b\b\b\b\b%c %4d", rotor[j%4], j+1);
      fflush(stdout);
      step(nu, mu, fi, ki, dt);
    }
    hipEventRecord(t1, 0);

    hipEventSynchronize(t1);
    hipEventElapsedTime(&ms, t0, t1); ms /= ns;
    printf("\b\b\b\b\b\b%.3f ms/cycle ~ %.3f GFLOPS\n",
           ms, 1e-6 * flop / ms);

    hipMemcpy(Host, W, sizeof(R), hipMemcpyDeviceToHost);
    if(Host[0].r == Host[0].r) /* spectrum is finite */
      dump(name(i), W);
    else if(exist(name(--i)) && load(W, name(i))) /* spectrum contains NAN */
      typical_v *= sqrt(2.0);
    else {
      fflush(stdout);
      fprintf(stderr, "diverged, fail to resume from \"%s\", QUIT\n", name(i));
      exit(-1);
    }
  }

  printf("======================= Done  Simulation =======================\n");

  hipEventDestroy(t1);
  hipEventDestroy(t0);
  return 0;
}
