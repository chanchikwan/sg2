#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "ihd.h"

int solve(R nu, R mu, R fi, R ki, R tt, Z i, Z n)
{
  const char rotor[] = "-/|\\";
  const R flop = 5 * N1 * N2 * (21.5 + (fi * ki > 0.0 ? 0 : 8) +
                                12.5 * (log2((double)N1) + log2((double)N2)));
  hipEvent_t t0, t1;
  hipEventCreate(&t0);
  hipEventCreate(&t1);

  printf("======================= Start Simulation =======================\n");

  for(++i; i <= n; ++i) {
    float ms;
    Z ns = (Z)ceil(tt / n / getdt(1.0, nu, mu)), j;
    R dt =         tt / n / ns;
    printf("%4d: %5.2f -> %5.2f, dt ~ %.0e:       ",
           i, dt * ns * (i-1), dt * ns * i, dt);
    srand(Seed);

    hipEventRecord(t0, 0);
    for(j = 0; j < ns; ++j) {
      printf("\b\b\b\b\b\b%c %4d", rotor[j%4], j+1);
      fflush(stdout);
      step(nu, mu, fi, ki, dt);
    }
    hipEventRecord(t1, 0);

    hipEventSynchronize(t1);
    hipEventElapsedTime(&ms, t0, t1); ms /= ns;
    printf("\b\b\b\b\b\b%.3f ms/cycle ~ %.3f GFLOPS\n",
           ms, 1e-6 * flop / ms);

    hipMemcpy(Host, W, sizeof(R), hipMemcpyDeviceToHost);
    if(Host[0].r != Host[0].r) {
      fflush(stdout);
      fprintf(stderr, "vorticity spectrum contains NAN, QUIT\n");
      exit(-1);
    }

    dump(name(i), W);
  }

  printf("======================= Done  Simulation =======================\n");

  hipEventDestroy(t1);
  hipEventDestroy(t0);
  return 0;
}
