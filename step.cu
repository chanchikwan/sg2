#include "hip/hip_runtime.h"
#include "ihd.h"

static __global__ void _evol_diff(C *f, const C *b, const R KK,
                                        const R nu, const R mu,
                                        const R im, const R ex,
                                        const Z n1, const Z h2)
{
  const Z i = blockDim.y * blockIdx.y + threadIdx.y;
  const Z j = blockDim.x * blockIdx.x + threadIdx.x;
  const Z h = i * h2 + j;

  if(i < n1 && j < h2) {
    const C g  = f[h];
    const C c  = b[h];
    const R kx = i < n1 / 2 ? i : i - n1;
    const R ky = j;
    const R kk = kx * kx + ky * ky;

    if(kk < KK) {
      const R imkk = im     * (nu * kk + mu);
      const R temp = K(1.0) / (K(1.0) + imkk);
      const R impl = temp   * (K(1.0) - imkk);
      const R expl = temp   * ex;

      f[h].r = impl * g.r + expl * c.r;
      f[h].i = impl * g.i + expl * c.i;
    } else {
      f[h].r = K(0.0);
      f[h].i = K(0.0);
    }
  }
}

void step(R nu, R mu, R fi, R ki, R dt)
{
  const R K = 0.99 + (MIN(N1, N2) - 1) / 3;

  const R alpha[] = {0.0,             0.1496590219993, 0.3704009573644,
                     0.6222557631345, 0.9582821306748, 1.0};
  const R beta[]  = {0.0,            -0.4178904745,   -1.192151694643,
                     -1.697784692471, -1.514183444257};
  const R gamma[] = {0.1496590219993, 0.3792103129999, 0.8229550293869,
                     0.6994504559488, 0.1530572479681};

  int i;
  for(i = 0; i < 5; ++i) {
    const R im = dt * 0.5 * (alpha[i+1] - alpha[i]);
    const R ex = dt * gamma[i] / (N1 * N2);

    if(fi * ki > 0.0)
      scale(w, beta[i]);
    else
      force(w, beta[i], fi, ki); /* scaling and Kolmogorov forcing */

    dx_dd_dy(X, Y, W); add_pro(w, inverse((R *)X, X), inverse((R *)Y, Y));
    dy_dd_dx(Y, X, W); sub_pro(w, inverse((R *)Y, Y), inverse((R *)X, X));

    forward(X, w); /* X here is just a buffer */

    _evol_diff<<<Hsz, Bsz>>>(W, (const C *)X, K * K, nu, mu, im, ex, N1, H2);
  }

  if(fi * ki > 0)
    force(W, dt, fi, ki); /* 1st-order Euler update for random forcing */
}
